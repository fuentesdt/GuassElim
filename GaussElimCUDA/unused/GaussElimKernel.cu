#include "hip/hip_runtime.h"
// Gaussian Elimination Function

__global__ void ForwardElimKernel(double d_A[3][4], double d_piv[3], nDim);

void GaussElim(double h_A[3][4], double h_b[3], double h_x[3], const int nDim)
{
	double d_A[3][4], d_piv[3];

	// Allocate memory on device
	//hipMalloc(d_A,sizeof(float)*(numvar)*(numvar+1));
	//hipMalloc(d_piv,sizeof(float)*(numvar)*(numvar+1));

	// Copy data from host to device
	//hipMemcpy(a_d, temp_h, sizeof(float)*numvar*(numvar+1),cudaMemcpyHostTo Device);

	// Define thread block size
	//dim3 dimBlock(numvar+1,numvar,1);
	//dim3 dimGrid(1,1,1);

	// Forward elimination kernel
	ForwardElimKernel<<<dimGrid , dimBlock>>>(d_A, d_piv, nDim);

	// Copy data from device to host
	//hipMemcpy(temp1_h,b_d,sizeof(float)*numvar*(numvar+1),cudaMemcpyDeviceT oHost);

	// Free memory on device
	hipFree(d_A);
	hipFree(d_piv);

	// Backward substitution
	for (int i = 0; nDim-1; i++)
		h_b[i] = h_A[i][nDim];	
	for (int i = nDim-1; i >= 0; i--)
	{
		h_x[i] = h_b[i];
		for (int j = nDim-1; j >= i+1; j--)
			h_x[i] = h_x[i] - h_A[i][j]*h_x[j];
		h_x[i] = h_x[i]/h_A[i][i];
	}
}

