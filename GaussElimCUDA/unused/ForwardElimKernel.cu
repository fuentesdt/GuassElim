
#include <hip/hip_runtime.h>
// Kernel for forward elimination in Gauss elimination

// #include <cuda.h>

__global__ void ForwardElimKernel(double *d_A[10][11], double *d_piv[10], int nDim)
{
	// Assign matrix elements to blocks and threads
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	// Parallel forward elimination
	for (int k = 0; k <= nDim-2; k++)
	{
		*d_piv[i] = *d_A[i][k]/(*d_A[k][k]);
		__syncthreads();
		if ((i>k) && (i<nDim) && (j>=k) && (j<=nDim))
			*d_A[i][j] -= *d_piv[i]*(*d_A[k][j]);
		__syncthreads();
	}
}
