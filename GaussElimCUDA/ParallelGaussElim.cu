
#include <hip/hip_runtime.h>

/*
 * Device code
 */

__global__ void ParallelGaussElim(
	int const nDim_image,
	int const nDim_matrix,
	double* d_A,
	double* d_b,
	double* d_x)
{
	// Assign image pixels to blocks and threads
	int i_image = blockDim.x*blockIdx.x + threadIdx.x;
	//int i_image = blockDim.y*blockIdx.y + threadIdx.y;
//printf("blockDim.x = %i \n",blockDim.x);
//printf("blockIdx.x = %i \n",blockIdx.x);
//printf("threadIdx.x = %i \n",threadIdx.x);
//printf("i_image = %i \n", i_image);
	//int offset = (j_image + i_image*nDim_image)*nDim_matrix*nDim_matrix;
	int offset_2d = i_image*nDim_matrix*nDim_matrix;
	int offset_1d = i_image*nDim_matrix;
//printf("offset = %i \n", offset);
	// Gauss elimination
	//int nDim_local = 8;
	//double local_A[nDim_local*nDim_local] = 0;
	for (int k=0; k<nDim_matrix-1; k++)
	{
		for (int i=k+1;	i<nDim_matrix; i++)
		{
			double pivot = d_A[offset_2d+i+k*nDim_matrix]/d_A[offset_2d+k+k*nDim_matrix];
			for (int j=k; j<nDim_matrix; j++)
			{
				d_A[offset_2d+i+j*nDim_matrix] -= pivot*d_A[offset_2d+k+j*nDim_matrix];
			}
			d_b[offset_1d+i] -= pivot*d_b[offset_1d+k];
		}
	}

	// Backward substitution
	for (int i=nDim_matrix-1; i>=0; i--)
	{
		d_x[offset_1d+i] = d_b[offset_1d+i];
		for (int j=nDim_matrix-1; j>i; j--)
		{
			d_x[offset_1d+i] -= d_A[offset_2d+i+j*nDim_matrix]*d_x[offset_1d+j];
		}
        	d_x[offset_1d+i] = d_x[offset_1d+i]/d_A[offset_2d+i+i*nDim_matrix];
if (d_x[offset_1d+i] /= 1) printf ("blkdim,id,tdid = %i,%i,%i \n",blockDim.x,blockIdx.x,threadIdx.x);
	}
}
